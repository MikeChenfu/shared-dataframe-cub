#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

 #include "cudf/utilities/type_dispatcher.hpp"

 #include <cudf/column/column_device_view.cuh>
 #include <cudf/column/column_factories.hpp>
 #include <cudf/detail/copy.hpp>
 #include <cudf/detail/gather.hpp>
 #include <cudf/detail/nvtx/ranges.hpp>
 #include <cudf/detail/utilities/cuda.cuh>
 #include <cudf/detail/utilities/device_atomics.cuh>
 #include <cudf/null_mask.hpp>
 #include <cudf/strings/string_view.cuh>
 #include <cudf/table/table.hpp>
 #include <cudf/table/table_view.hpp>
 #include <cudf/types.hpp>
 #include <cudf/utilities/default_stream.hpp>
 #include <cudf/utilities/error.hpp>
 #include <cudf/utilities/traits.hpp>
 //#include <cudf/utilities/type_dispatcher.hpp>
 
 #include "kernel_wrapper.hpp"
 #include <hipcub/hipcub.hpp>
 #include <cub/util_allocator.cuh>
 #include <cub/device/device_scan.cuh>
 #include <cub/device/device_select.cuh>
 #include <cub/device/device_radix_sort.cuh>
 #include <cub/device/device_run_length_encode.cuh>
 
 #include <rmm/cuda_stream_view.hpp>
 #include <rmm/device_buffer.hpp>
 #include <rmm/device_scalar.hpp>
 #include <rmm/device_uvector.hpp>
 #include <rmm/exec_policy.hpp>
 
 
 #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
 #define DEF_THREAD_CNT 96
 inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
 {
    if (code != hipSuccess) 
    {
       fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
       if (abort) exit(code);
    }
 }
 __global__ void initOffsets( int total, int seg_len, uint32_t *out)
 {
    int idx=blockIdx.x*blockDim.x+threadIdx.x;
 
    if(idx<total)
    {
       out[idx]=seg_len;
    }
    return;
 }
 
 int getMaxMeanSales( float *d_in_array, int in_len, int seg_len, float **d_out_array)
 {
     //hipStream_t stream1, stream2, stream3, stream4 ;
 
     //hipStreamCreate ( &stream2) ;
 
     size_t bytes_scan=0;
     size_t temp_storage_bytes=0;
     uint32_t *d_A;
     uint32_t *d_offsets;
     void *d_temp = NULL;
 
 
     gpuErrchk (hipMalloc (
         (void **) &d_A,
         (in_len+2 + DEF_THREAD_CNT) * sizeof (uint32_t)
     ));
 
     gpuErrchk (hipMalloc (
         (void **) &d_offsets,
         (in_len+2 + DEF_THREAD_CNT) * sizeof (uint32_t)
     ));
 /*
     gpuErrchk (hipMalloc (
         (void **) d_out_array,
         (in_len/seg_len) * sizeof (int)
     ));
  */
     printf("sim1\n");
     initOffsets<<<(in_len+1)/DEF_THREAD_CNT+1,DEF_THREAD_CNT>>>(in_len+1, seg_len, d_A);
 
     gpuErrchk (hipDeviceSynchronize ());
 
 
     gpuErrchk (hipcub::DeviceScan::ExclusiveSum (
             NULL, bytes_scan, d_A, d_offsets, in_len
     ));
     gpuErrchk (hipDeviceSynchronize ());
 
     gpuErrchk (hipMalloc (&d_temp, bytes_scan));
     //rmm::cuda_stream_view stream        = cudf::default_stream_value;
     //rmm::device_buffer d_temp_storage(bytes_scan,stream);
 
     gpuErrchk (hipcub::DeviceScan::ExclusiveSum (
             d_temp, bytes_scan, d_A, d_offsets, in_len
     ));
    
     
     printf("sim2\n");
     //printValues<<<(10)/DEF_THREAD_CNT+1,DEF_THREAD_CNT>>>(10, d_A, d_offsets);
     //printValues<<<(in_len+1)/DEF_THREAD_CNT+1,DEF_THREAD_CNT>>>(in_len+1, d_A, d_offsets);
     
     //gpuErrchk (hipDeviceSynchronize ());
     printf("3\n");
 
    
     gpuErrchk (hipFree (d_temp));
     gpuErrchk (hipFree (d_A));
     gpuErrchk (hipFree (d_offsets));
 
 
     return hipSuccess;
 
 }
 
 static constexpr double mm_to_inches = 0.0393701;
 
 __global__ void kernel_tenth_mm_to_inches(cudf::mutable_column_device_view val, cudf::mutable_column_device_view data, cudf::mutable_column_device_view res_data, cudf::mutable_column_device_view res_val)
 {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i < val.size()) {
       res_val.element<double>(i) = val.element<double>(i) * mm_to_inches;
       res_data.element<int>(i) = data.element<int>(i) + 1;
     }
 }
  
  CudfWrapper::CudfWrapper(cudf::mutable_table_view table_view, cudf::mutable_table_view result) {
  
   mtv = table_view;
    res = result;
 
  }
  
  void CudfWrapper::tenth_mm_to_inches() {
 
 
   // Example of showing num_columns and num_rows only for potential debugging
   printf("kernel_wrapper.cu input: # of columns: %lu\n", mtv.num_columns());
   printf("kernel_wrapper.cu input: # of rows: %lu\n", mtv.num_rows());
   printf("kernel_wrapper.cu output: # of columns: %lu\n", res.num_columns());
   printf("kernel_wrapper.cu output: # of rows: %lu\n", res.num_rows());
  /* 
   // get the target columns from the table and index starting from 1
   // e.g. ['station_id','data', 'type','val'], the index of val is 4.
   std::unique_ptr<cudf::mutable_column_device_view, std::function<void(cudf::mutable_column_device_view*)>> 
        val = cudf::mutable_column_device_view::create(mtv.column(4));
         
   std::unique_ptr<cudf::mutable_column_device_view, std::function<void(cudf::mutable_column_device_view*)>> 
        data = cudf::mutable_column_device_view::create(mtv.column(2));
 
   // get the result columns from the table
   std::unique_ptr<cudf::mutable_column_device_view, std::function<void(cudf::mutable_column_device_view*)>> 
        res_val = cudf::mutable_column_device_view::create(res.column(1));
         
   std::unique_ptr<cudf::mutable_column_device_view, std::function<void(cudf::mutable_column_device_view*)>> 
        res_data = cudf::mutable_column_device_view::create(res.column(2));
 
   // Invoke the Kernel to convert tenth_mm -> inches
   hipStream_t stream1, stream2, stream3, stream4 ;
   hipStreamCreate ( &stream1) ;
   kernel_tenth_mm_to_inches<<<(mtv.num_rows()+255)/256, 256>>>(*val, *data, *res_data, *res_val);
   hipError_t err = hipStreamSynchronize(0);
   printf("hipStreamSynchronize Response = %d\n", (int)err);
 
   */ 
   float *d_wkly_mean = NULL;
   float *d_max_mean=NULL;
   getMaxMeanSales( d_wkly_mean, 3000*52, 52, &d_max_mean );
   hipError_t err = hipStreamSynchronize(0);
   printf("hipStreamSynchronize Response = %d\n", (int)err);
   
  
  }
  
  CudfWrapper::~CudfWrapper() {
    // It is important to note that CudfWrapper does not own the underlying Dataframe 
    // object and that will be freed by the Python/Cython layer later.
  }
 